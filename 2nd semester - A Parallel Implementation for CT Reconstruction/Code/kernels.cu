#include "hip/hip_runtime.h"
#include "kernels.h"
#include <iostream>
#include <cmath>
#include <iostream>
#include <sys/time.h>
#include <unistd.h>
#include "get_micro_second.h"
#include <cstdlib>


// Includes CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>


// 2D float texture
texture<float, hipTextureType2D, hipReadModeElementType> texRef;



/*
 * Function Name : sinogram_serial
 ***************************************************************************
 * This fucntion takes in the filtered sinogram and backprojects it. BackProjection
 * is calculated using the formula of inverse radon transform.
 ***************************************************************************
 * angles: int : total number of angles in the sinogram
 * sensors : int : number of sensors in the sinogram.
 * theta: float *: the values of theta at which the image projections are required
 		   in the sinogram
 * x_min: float : this is minimum pixel distance in the x direction
 * y_min: float : this is the minimum pixel distance in the y direction
 * r_min: float : this is the minimum pixel distance along the diagonal direction
 * dx: float : the x pixel width
 * dy: float: the y pixel width
 * dr: float : the pixel diagonal length
 * Width: int : the width of the original image to be reconstructed
 * Height: int: the height of the original image to be reconstructed
 * img: float *: the image whose sinogram needs to be calculated
 * sinogram_output: float *: the sinogram that has been calculated
 ***************************************************************************
 * Returns void
 */

void sinogram_serial(float * img,float* sinogram_output, float dx, float x_min, float dy, float y_min, int sensors, float dr, float r_min, int angles, float* theta,int Width,int Height)
{
    for (int sensor_no=0 ; sensor_no < sensors ; sensor_no++)
    {
    	for (int angle_no=0; angle_no <angles ; angle_no++)
    	{

		    if (sensor_no < sensors && angle_no < angles) 
		    {
		        float sum = 0;
		        float r = sensor_no * dr + r_min;
		        int ind_x,ind_y;
		        float d00,d11,d10,d01;
		        float a,b;
		        float result_temp1,result_temp2;
		        for (int z_idx = 0; z_idx < sensors; z_idx++) 
		        {
		            float z = z_idx * dr + r_min;


		            // Transform coordinates------from r, theta to x, t-----------------------------------------------
		            float r_real = (r * cosf(theta[angle_no]) + z * sinf(theta[angle_no]) - x_min)/dx + 0.5f;
		            float z_real = (z * cosf(theta[angle_no]) - r * sinf(theta[angle_no]) - y_min)/dy + 0.5f;

            		//BILINEAR INTERPOLATION START
	               if ((r_real<Width)&&(z_real<Height)) 
	               {

	     

	                   ind_x = floor(r_real);
	                   a      = r_real-ind_x;

	                   ind_y = floor(z_real);
	                   b      = z_real-ind_y;

	                   if (((ind_x)   < Width)&&((ind_y)   < Height))    d00 = img[ind_y*Height+ind_x];   else d00 = 0;     
	                   if (((ind_x+1) < Width)&&((ind_y)   < Height))    d10 = img[ind_y*Height+ind_x+1]; else d10 = 0;      
	                   if (((ind_x)   < Width)&&((ind_y+1) < Height))    d01 = img[(ind_y+1)*Height+ind_x];   else d01 = 0; 
	                   if (((ind_x+1) < Width)&&((ind_y+1) < Height))    d11 = img[(ind_y+1)*Height+ind_x+1]; else d11 = 0;

	                    result_temp1 = a * d10+ (-d00 * a + d00);
	      

	                    result_temp2 = a * d11 + (-d01 * a + d01);
	                    sum += b * result_temp2 + (-result_temp1 * b + result_temp1);
	      
	                }
           
            
        		}
        		sinogram_output[angle_no*sensors + sensor_no] = sum;

			}
		}
	}
}


/*
 * Kernel Name : sinogram_kernel_tex
 ***************************************************************************
 * This kernel takes in the filtered sinogram and backprojects it. BackProjection
 * is calculated using the formula of inverse radon transform.
 ***************************************************************************
 * angles: int : total number of angles in the sinogram
 * sensors : int : number of sensors in the sinogram.
 * theta: float *: the values of theta at which the image projections are required
 		   in the sinogram
 * x_min: float : this is minimum pixel distance in the x direction
 * y_min: float : this is the minimum pixel distance in the y direction
 * r_min: float : this is the minimum pixel distance along the diagonal direction
 * dx: float : the x pixel width
 * dy: float: the y pixel width
 * dr: float : the pixel diagonal length
 * Width: int : the width of the original image to be reconstructed
 * Height: int: the height of the original image to be reconstructed
 * img: float *: the image whose sinogram needs to be calculated
 * sinogram_output: float *: the sinogram that has been calculated
 ***************************************************************************
 * Returns void
 */

__global__ void sinogram_kernel_tex(float * img,float* sinogram_output, float dx, float x_min, float dy, float y_min, int sensors, float dr, float r_min, int angles, float* theta,int Width,int Height)
{
    unsigned int sensor_no = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int angle_no = blockIdx.y * blockDim.y + threadIdx.y;

    if (sensor_no < sensors && angle_no < angles) 
    {
        float sum = 0;
        float r = sensor_no * dr + r_min;
        for (int z_idx = 0; z_idx < sensors; z_idx++) 
        {
            float z = z_idx * dr + r_min;


            // Transform coordinates------from r, theta to x, t-----------------------------------------------
            float r_real = (r * cosf(theta[angle_no]) + z * sinf(theta[angle_no]) - x_min)/dx + 0.5f;
            float z_real = (z * cosf(theta[angle_no]) - r * sinf(theta[angle_no]) - y_min)/dy + 0.5f;

            sum += tex2D(texRef, r_real, z_real);
      

        }

        sinogram_output[angle_no*sensors + sensor_no] = sum;
    }
}


/*
 * Kernel Name : sinogram_kernel
 ***************************************************************************
 * This kernel takes in the filtered sinogram and backprojects it. BackProjection
 * is calculated using the formula of inverse radon transform.
 ***************************************************************************
 * angles: int : total number of angles in the sinogram
 * sensors : int : number of sensors in the sinogram.
 * theta: float *: the values of theta at which the image projections are required
 		   in the sinogram
 * x_min: float : this is minimum pixel distance in the x direction
 * y_min: float : this is the minimum pixel distance in the y direction
 * r_min: float : this is the minimum pixel distance along the diagonal direction
 * dx: float : the x pixel width
 * dy: float: the y pixel width
 * dr: float : the pixel diagonal length
 * Width: int : the width of the original image to be reconstructed
 * Height: int: the height of the original image to be reconstructed
 * img: float *: the image whose sinogram needs to be calculated
 * sinogram_output: float *: the sinogram that has been calculated
 ***************************************************************************
 * Returns void
 */

__global__ void sinogram_kernel(float * img,float* sinogram_output, float dx, float x_min, float dy, float y_min, int sensors, float dr, float r_min, int angles, float* theta,int Width,int Height)
{
    unsigned int sensor_no = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int angle_no = blockIdx.y * blockDim.y + threadIdx.y;

    if (sensor_no < sensors && angle_no < angles) 
    {
        float sum = 0;
        float r = sensor_no * dr + r_min;
        int ind_x,ind_y;
        float d00,d11,d10,d01;
        float a,b;
        float result_temp1,result_temp2;
        for (int z_idx = 0; z_idx < sensors; z_idx++) 
        {
            float z = z_idx * dr + r_min;


            // Transform coordinates------from r, theta to x, t-----------------------------------------------
            float r_real = (r * cosf(theta[angle_no]) + z * sinf(theta[angle_no]) - x_min)/dx + 0.5f;
            float z_real = (z * cosf(theta[angle_no]) - r * sinf(theta[angle_no]) - y_min)/dy + 0.5f;
            
            //BILINEAR INTERPOLATION START
               if ((r_real<Width)&&(z_real<Height)) 
               {

     

                   ind_x = floor(r_real);
                   a      = r_real-ind_x;

                   ind_y = floor(z_real);
                   b      = z_real-ind_y;

                   if (((ind_x)   < Width)&&((ind_y)   < Height))    d00 = img[ind_y*Height+ind_x];   else d00 = 0;     
                   if (((ind_x+1) < Width)&&((ind_y)   < Height))    d10 = img[ind_y*Height+ind_x+1]; else d10 = 0;      
                   if (((ind_x)   < Width)&&((ind_y+1) < Height))    d01 = img[(ind_y+1)*Height+ind_x];   else d01 = 0; 
                   if (((ind_x+1) < Width)&&((ind_y+1) < Height))    d11 = img[(ind_y+1)*Height+ind_x+1]; else d11 = 0;

                    result_temp1 = a * d10+ (-d00 * a + d00);
      

                    result_temp2 = a * d11 + (-d01 * a + d01);
                    sum += b * result_temp2 + (-result_temp1 * b + result_temp1);
      
                }
           
            //  BILINEAR INTERPOLATION END
            
        }

        sinogram_output[angle_no*sensors + sensor_no] = sum;
    }
}


/*
 * Function Name : sinogram
 ***************************************************************************
 * This function calls the sinogram kernel and returns the sinogram calculated
 * This function is also responsible for memory allocation and resource freeing
 * for calling the concerned kernel
 ***************************************************************************
 * angles: int : total number of angles in the sinogram
 * sensors : int : number of sensors in the sinogram.
 * theta: float *: the values of theta at which the image projections are required
 		   in the sinogram
 * x_min: float : this is minimum pixel distance in the x direction
 * y_min: float : this is the minimum pixel distance in the y direction
 * r_min: float : this is the minimum pixel distance along the diagonal direction
 * dx: float : the x pixel width
 * dy: float: the y pixel width
 * dr: float : the pixel diagonal length
 * Width: int : the width of the original image to be reconstructed
 * Height: int: the height of the original image to be reconstructed
 * img: float *: the image whose sinogram needs to be calculated
 * h_img_out_t: float **: the sinogram output is returned
 ***************************************************************************
 * Returns void
 */


void sinogram (int angles, float * theta, int Width, int Height, float * img , int sensors,float dx,float dy, float dr,float x_min,float y_min,float r_min, float **h_img_out_t)
{
    //
    // Declare the variables for measuring elapsed time
    double sTime;
    double eTime;
    
    float* device_angles,*device_img;
    //Allocating space 
    hipMalloc(&device_angles, angles * sizeof(float));
    hipMalloc(&device_img, Width * Height * sizeof(float));

    sTime = getMicroSecond();

    // Copy host to device
    hipMemcpy(device_angles, theta, angles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_img, img, Width * Height * sizeof(float), hipMemcpyHostToDevice);

    eTime = getMicroSecond();
    double SinogramTransferTime = eTime - sTime;

    // Allocate result of sinogram kernel in device memory
    float* device_result;
    hipMalloc(&device_result, sensors * angles * sizeof(float));

    // Kernel Initializations
    dim3 dimBlock(16, 16, 1); // 256 threads per block
    dim3 dimGrid((sensors + dimBlock.x - 1) / dimBlock.x, (angles + dimBlock.y - 1) / dimBlock.y, 1); // rounding of to the next int according to image dimensions
    sTime = getMicroSecond();

    // Calling the Kernel
    sinogram_kernel<<<dimGrid, dimBlock>>>(device_img,device_result, dx, x_min, dy, y_min, sensors, dr, r_min, angles, device_angles,Width,Height);

    eTime = getMicroSecond();

    double singogramKernelTime = eTime - sTime;
    std::cout <<"Sinogram Kernel Time = "<< singogramKernelTime * 1e3 << "[ms]" <<std::endl;

    // Returning the image back
    float *h_img_out;
    h_img_out = (float *)malloc(sensors * angles * sizeof(float));

    /*
    sTime = getMicroSecond();

    sinogram_serial(img, h_img_out,  dx,  x_min,  dy, y_min, sensors, dr, r_min,  angles, theta, Width, Height);

    eTime = getMicroSecond();
    SinogramTransferTime += ( eTime - sTime );
    std::cout <<"Sinogram Serial time is = "<< SinogramTransferTime * 1e3 << "[ms]" <<std::endl;
    */
    
    sTime = getMicroSecond();

    // Copy Device to Host
    hipMemcpy(h_img_out, device_result, sensors * angles * sizeof(float), hipMemcpyDeviceToHost);

    eTime = getMicroSecond();
    SinogramTransferTime += ( eTime - sTime );
    std::cout <<"Sinogram Data Transfer Time = "<< SinogramTransferTime * 1e3 << "[ms]" <<std::endl;
    
    for( unsigned int i = 0; i < angles; i++ ) 
    {
    	for( unsigned int j = 0; j < sensors; j++ )
    	{
        	unsigned int pixelPos = i * sensors + j;
        	(*h_img_out_t)[pixelPos] = (h_img_out)[pixelPos];
    	}
    }
    // Free device memory
    hipFree(device_result);

}

/*
 * Function Name : sinogram_tex
 ***************************************************************************
 * This function calls the sinogram kernel and returns the sinogram calculated
 * This function is also responsible for memory allocation and resource freeing
 * for calling the concerned kernel
 ***************************************************************************
 * angles: int : total number of angles in the sinogram
 * sensors : int : number of sensors in the sinogram.
 * theta: float *: the values of theta at which the image projections are required
 		   in the sinogram
 * x_min: float : this is minimum pixel distance in the x direction
 * y_min: float : this is the minimum pixel distance in the y direction
 * r_min: float : this is the minimum pixel distance along the diagonal direction
 * dx: float : the x pixel width
 * dy: float: the y pixel width
 * dr: float : the pixel diagonal length
 * Width: int : the width of the original image to be reconstructed
 * Height: int: the height of the original image to be reconstructed
 * img: float *: the image whose sinogram needs to be calculated
 * h_img_out_t: float **: the sinogram output is returned
 ***************************************************************************
 * Returns void
 */


void sinogram_tex(int angles, float * theta, int Width, int Height, float * img , int sensors,float dx,float dy, float dr,float x_min,float y_min,float r_min, float **h_img_out_t)
{
    //
    // Declare the variables for measuring elapsed time
    double sTime;
    double eTime;
    
    float* device_angles,*device_img;
    //Allocating space 
    hipMalloc(&device_angles, angles * sizeof(float));
    hipMalloc(&device_img, Width * Height * sizeof(float));

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, Width, Height);

    // Copy to device memory some data located at address h_img in host memory 
    hipMemcpyToArray(cuArray, 0, 0, img, Width * Height * sizeof(float), hipMemcpyHostToDevice);

    // Set texture reference parameters
    texRef.addressMode[0] = hipAddressModeBorder;
    texRef.addressMode[1] = hipAddressModeBorder;
    texRef.filterMode = hipFilterModeLinear;
    texRef.normalized = false;

    // Bind the array to the texture reference
    hipBindTextureToArray(texRef, cuArray, channelDesc);

    sTime = getMicroSecond();

    // Copy host to device
    hipMemcpy(device_angles, theta, angles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_img, img, Width * Height * sizeof(float), hipMemcpyHostToDevice);

    eTime = getMicroSecond();
    double SinogramTransferTime = eTime - sTime;

    // Allocate result of sinogram kernel in device memory
    float* device_result;
    hipMalloc(&device_result, sensors * angles * sizeof(float));

    // Kernel Initializations
    dim3 dimBlock(16, 16, 1); // 256 threads per block
    dim3 dimGrid((sensors + dimBlock.x - 1) / dimBlock.x, (angles + dimBlock.y - 1) / dimBlock.y, 1); // rounding of to the next int according to image dimensions
    sTime = getMicroSecond();

    // Calling the Kernel
    sinogram_kernel<<<dimGrid, dimBlock>>>(device_img,device_result, dx, x_min, dy, y_min, sensors, dr, r_min, angles, device_angles,Width,Height);

    eTime = getMicroSecond();

    double singogramKernelTime = eTime - sTime;
    std::cout <<"Sinogram Kernel Time = "<< singogramKernelTime * 1e3 << "[ms]" <<std::endl;

    // Returning the image back
    float *h_img_out;
    h_img_out = (float *)malloc(sensors * angles * sizeof(float));

    sTime = getMicroSecond();

    // Copy Device to Host
    hipMemcpy(h_img_out, device_result, sensors * angles * sizeof(float), hipMemcpyDeviceToHost);

    eTime = getMicroSecond();
    SinogramTransferTime += ( eTime - sTime );
    std::cout <<"Sinogram Data Transfer Time = "<< SinogramTransferTime * 1e3 << "[ms]" <<std::endl;

    for( unsigned int i = 0; i < angles; i++ ) 
    {
    	for( unsigned int j = 0; j < sensors; j++ )
    	{
        	unsigned int pixelPos = i * sensors + j;
        	(*h_img_out_t)[pixelPos] = (h_img_out)[pixelPos];
    	}
    }
    // Free device memory
    hipFree(device_result);

}


/*
 * Kernel Name : filterationkernel
 ***************************************************************************
 * This kernel takes in the fft of the sinogram and multiplies with the
 * ramlak filter. This is intended to suppress the low frequencies and intensify 
 * the high frequency content.
 ***************************************************************************
 * filter_subject: hipfftComplex* : This the fft of the sinogram that needs to be
 				   filtered.
 * sensors : int : number of sensors in the sinogram.
 * angles : int : number of angles in the sinogram.
 ***************************************************************************
 * Returns void
 */

__global__ void filterationkernel(hipfftComplex* filter_subject, int sensors, int angles)
{
    unsigned int sensor_no = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int angle_no = blockIdx.y * blockDim.y + threadIdx.y;

    if (sensor_no < sensors && angle_no < angles)
    {
        filter_subject[sensor_no + sensors*angle_no].x *= ((sensor_no< sensors - sensor_no) ? sensor_no :(sensors - sensor_no)) / (float)sensors;
        filter_subject[sensor_no + sensors*angle_no].y *= ((sensor_no< sensors - sensor_no) ? sensor_no :(sensors - sensor_no)) / (float)sensors;
    }
}



/*
 * Kernel Name : inversefft_real
 ***************************************************************************
 * This kernel takes in the inverse fft of the filtered sinogram and returns
 * only the real part of the inverse fft.
 ***************************************************************************
 * filter_subject: hipfftComplex* : This the fft of the sinogram that needs to be
 				   filtered.
 * sensors : int : number of sensors in the sinogram.
 * angles : int : number of angles in the sinogram.
 ***************************************************************************
 * Returns void
 */

 __global__ void inversefft_real(float* real_ifft, hipfftComplex* ifft, int len_ifft)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len_ifft)  
    {
    	real_ifft[index] = ifft[index].x;
   	}
}

/*
 * Kernel Name : backprojection_kernel
 ***************************************************************************
 * This kernel takes in the filtered sinogram and backprojects it. BackProjection
 * is calculated using the formula of inverse radon transform.
 ***************************************************************************
 * angles: int : total number of angles in the sinogram
 * sensors : int : number of sensors in the sinogram.
 * theta: float *: the total number of thetas used.
 * x_min: float : this is minimum pixel distance in the x direction
 * y_min: float : this is the minimum pixel distance in the y direction
 * r_min: float : this is the minimum pixel distance along the diagonal direction
 * dx: float : the x pixel width
 * dy: float: the y pixel width
 * dr: float : the pixel diagonal length
 * Width: int : the width of the original image to be reconstructed
 * Height: int: the height of the original image to be reconstructed
 * filtered_sinogram: float *: filtered sinogram of the image
 * output_recon: float *: this is the reconstructed image that is to be used
 ***************************************************************************
 * Returns void
 */

 __global__ void backprojection_kernel(int angles,int sensors, float *theta,float x_min, float dx, int Width,float y_min, float dy, int Height,float r_min, float dr, float *output_recon,float *filtered_sinogram)
{
    unsigned int x_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_index = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_index < Width && y_index < Height)
    {
        float x, y, unscaled_r;
        float sum = 0;
        int sensor_index;

        x = x_min + x_index * dx;
        y = y_min + y_index * dy;

        for (int theta_idx = 0; theta_idx < angles; theta_idx++) 
        {
            unscaled_r = x*cosf(theta[theta_idx] * M_PI / 180.0f) + y*sinf(theta[theta_idx] * M_PI / 180.0f);
            sensor_index = (unscaled_r - r_min) / dr ; 
            sum +=  filtered_sinogram[theta_idx*sensors + sensor_index];
        }
        output_recon[x_index + Width * y_index] = sum;
    }
}






/*
 * Function Name : backProjection
 ***************************************************************************
 * This function calls the sinogram kernel and returns the sinogram calculated
 * This function is also responsible for memory allocation and resource freeing
 * for calling the concerned kernel
 ***************************************************************************
 * angles: int : total number of angles in the sinogram
 * sensors : int : number of sensors in the sinogram.
 * theta: float *: the values of theta at which the image projections are required
 		   in the sinogram
 * x_min: float : this is minimum pixel distance in the x direction
 * y_min: float : this is the minimum pixel distance in the y direction
 * r_min: float : this is the minimum pixel distance along the diagonal direction
 * dx: float : the x pixel width
 * dy: float: the y pixel width
 * dr: float : the pixel diagonal length
 * Width: int : the width of the original image to be reconstructed
 * Height: int: the height of the original image to be reconstructed
 * sinogram_image: float *: the sinogram whose output needs to be calculated
 * reconstructed_out: float **: the reconstructed image is returned 
 * filter_out: float **: Output of the filter that is to be used
 ***************************************************************************
 * Returns void
 */

int backProjection (float ** filter_out ,float * sinogram_image, int angles, float * theta, int Width, int Height, int sensors,float dx,float dy, float dr,float x_min,float y_min,float r_min, float **reconstructed_out)
{   

    // Declare the variables for measuring elapsed time
    double sTime;
    double eTime;

    

/********************************************* FILTERATION STARTS ************************************************************/
    // Declaring the hipfftComplex Variable on the Host side + fft_input variable of the image.
    hipfftComplex* fft_input;
    fft_input = (hipfftComplex *)malloc(sizeof(hipfftComplex) * angles * sensors);
    float *filtered_result;
    filtered_result = (float*)malloc(angles*sensors*sizeof(float));

   	for( unsigned int i = 0; i < angles; i++ ) 
    {
        for( unsigned int j = 0; j < sensors; j++ ) 
        {
           unsigned int pixelPos = i * sensors + j;
           fft_input[pixelPos].x = sinogram_image[pixelPos];
           fft_input[pixelPos].y = sinogram_image[pixelPos];

        }
    }
  

     // Allocate space on the GLOBAL memory for theta and the filtered image
    sTime = getMicroSecond();
    float *device_theta, *device_filtered;
    hipMalloc(&device_theta, angles * sizeof(float));
    hipMalloc(&device_filtered, angles * sensors * sizeof(float));
    hipMemcpy(device_theta, theta, angles * sizeof(float), hipMemcpyHostToDevice);
    eTime = getMicroSecond();
    double BackProjectionTransferTime = eTime - sTime;


 /************************************** FFT OF THE SINOGRAM *********************/
    // FFT initialization to contain the fft of the sinogram
    hipfftComplex* device_fft;
    hipMalloc((void **)&device_fft, sizeof(hipfftComplex) * angles * sensors);


    // Copy the image sinogram to the fft
    sTime = getMicroSecond();
    hipMemcpy(device_fft, fft_input, sizeof(hipfftComplex)* angles * sensors, hipMemcpyHostToDevice);
    eTime = getMicroSecond();
    double filteringTransferTime = eTime - sTime;

    // hipfftHandle is used so that the configuration of the fft is used again and again
    // This reduces the overhead time
    hipfftHandle plan;
    hipfftPlan1d(&plan, sensors, HIPFFT_C2C, angles);

    // Execute FFT - Complex input, Complex output. We are overriding the input with the fft result
    hipfftExecC2C(plan, device_fft, device_fft, HIPFFT_FORWARD);


/***** FILTERATION OF THE SINOGRAM FFT WITH RAMLAK FILTER ***********************/
    // Now Ramp Filter the FFT
    dim3 dimBlockRF(16, 16, 1); // 256 Threads
    // rounding of to the next int according to image dimensions
    dim3 dimGridRF((sensors + dimBlockRF.x - 1) / dimBlockRF.x, 
        (angles + dimBlockRF.y - 1) / dimBlockRF.y, 1);

    sTime = getMicroSecond();
    filterationkernel << <dimGridRF, dimBlockRF >> >(device_fft, sensors, angles);
    eTime = getMicroSecond();
    double FilteringKernelTime = eTime - sTime;


/******************************** INVERSE FFT *********************************/
	hipfftExecC2C(plan, device_fft, device_fft, HIPFFT_BACKWARD);



/****************************** REAL PART OF THE FFT RESULT ******************/
    // Write the real part of output as the ramp filtered sinogram
    int thdsPerBlk = 256;
    int blksPerGrid = (sensors*angles + thdsPerBlk - 1) / thdsPerBlk;
 	sTime = getMicroSecond();
    inversefft_real << <blksPerGrid, thdsPerBlk >> >(device_filtered, device_fft, sensors*angles);
    eTime = getMicroSecond();
    FilteringKernelTime += (eTime - sTime);
    std::cout <<"Filtering Kernel Time = "<< FilteringKernelTime * 1e3 << "[ms]" <<std::endl;
    sTime = getMicroSecond();
    hipMemcpy(filtered_result, device_filtered, sizeof(float)*sensors*angles, hipMemcpyDeviceToHost);
    eTime = getMicroSecond();
    filteringTransferTime += (eTime - sTime);
    std::cout <<"Filtering Data Transfer Time = "<< filteringTransferTime * 1e3 << "[ms]" <<std::endl;

  	for( unsigned int i = 0; i < angles; i++ ) 
    {
        for( unsigned int j = 0; j < sensors; j++ ) 
        {
            unsigned int pixelPos = i * sensors + j;
            (*filter_out)[pixelPos] = (filtered_result)[pixelPos];
       
        }
    }
 



/****************************** BACK PROJECTION ******************************/
   

    // Allocate result of backprojection in device memory ----------------------------------------------
    float *d_output;
    hipMalloc(&d_output, Width * Height * sizeof(float));
    float *h_output;
    h_output = (float*)malloc(Width*Height*sizeof(float));

    // Invoke kernel to BackProject------------Kernel 3-------------------------------------------------------------
    dim3 dimBlockbackproj(16, 16, 1);
    dim3 dimGridbackproj((Width + dimBlockbackproj.x - 1) / dimBlockbackproj.x, (Height + dimBlockbackproj.y - 1) / dimBlockbackproj.y, 1);

    sTime = getMicroSecond();
    backprojection_kernel << <dimGridbackproj, dimBlockbackproj >>> (angles,sensors, device_theta, x_min, dx, Width, y_min, dy, Height, r_min, dr, d_output,device_filtered);
    eTime = getMicroSecond();
    double backProjectionKernelTime = eTime - sTime;
    std::cout <<"Back-Projection Kernel Time = "<< backProjectionKernelTime * 1e3 << "[ms]" <<std::endl;

    sTime = getMicroSecond();
    hipMemcpy(h_output, d_output, Width * Height * sizeof(float), hipMemcpyDeviceToHost);
    eTime = getMicroSecond();
    BackProjectionTransferTime += (eTime - sTime);
    std::cout <<"Back-Projection Data Transfer Time = "<< BackProjectionTransferTime * 1e3 << "[ms]" <<std::endl;


    // Returning the output
    for (int y_idx = 0; y_idx < Height; y_idx++) 
    {
        for (int x_idx = 0; x_idx < Width; x_idx++) 
        {
            (*reconstructed_out)[x_idx + Width * y_idx] = (h_output)[x_idx + Width * y_idx];
        }
    }
    return 0;
}

